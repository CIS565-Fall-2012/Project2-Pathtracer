#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com
#include <thrust\count.h>
#include <thrust\remove.h>
#include <thrust\device_vector.h>
#include <thrust\device_ptr.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include <cutil_math.h>
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#define traceDepth 10

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	ray r;
	r.origin = eye;

	glm::vec3 AVEC,BVEC,MVEC,HVEC,VVEC,Ppoint;//from CIS560 
	float Sx = x / (resolution.x );
	float Sy = y / (resolution.y );
	
	AVEC = glm::cross(view, up);//view is the CVEC, up is UVEC
	BVEC = glm::cross(AVEC, view);
	MVEC = eye + view;//Midpoint of screen
	HVEC =  view.length() * tan(fov.x) * glm::normalize(AVEC); 
	VVEC =  view.length() * tan(fov.y) * glm::normalize(BVEC);
	Ppoint = MVEC + ( 2*Sx - 1 ) * HVEC + ( 2*Sy -1 ) * VVEC; 
	
	r.direction = glm::normalize(Ppoint - eye);
	r.continueFlag = true;

	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//generate rays for further ray tracing
__global__ void generateRay(ray *rays, cameraData cam, float iter, float focus)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam.resolution.x);
	
	//for anti-aliasing
	thrust::default_random_engine rng( hash(index * iter) );
	thrust::uniform_real_distribution<float> X(-0.5, 0.5);
	float u = X(rng);
	float v = X(rng);
	
	if(x <= cam.resolution.x && y <= cam.resolution.y)
	{
		//rays[index] = raycastFromCameraKernel(cam.resolution, 0.0f, x, y, cam.position, cam.view, cam.up, cam.fov);
		// ///////////////////////////////////////////////////////////////////////////////////////
		//Anti-aliasing
		rays[index] = raycastFromCameraKernel(cam.resolution, 0.0f, x+u, y+v, cam.position, cam.view, cam.up, cam.fov); 
		rays[index].pixelId = index;
	}
	
	//////////////////////////////////////////////////////////////////////////////////////////////
	//DEPTH FIELD
	glm::vec3 FOC = rays[index].origin + rays[index].direction * focus;
	thrust::uniform_real_distribution<float> Y(-0.4, 0.4);
	float offsetX = Y(rng), offsetY = Y(rng);
	rays[index].origin += glm::vec3(offsetX, offsetY, 0.0f);
	rays[index].direction = glm::normalize(FOC - rays[index].origin);
	/////////////////////////////////////////////////////////////////////////////////////////

}

//This function returns the closest Geometry's ID
__host__ __device__ int getClosestGeom(staticGeom* geoms, int numberOfGeoms, ray &r, glm::vec3 &intersecP, glm::vec3 &normal)
{
	int closestGeomIndex = -1;
	float min_d = 10000;
	float d;//distance to the nearest geometry
	
	for( int i = 0; i < numberOfGeoms; ++i )
	{//this loop find out the closest object and light source
		if(geoms[i].type == SPHERE) d = sphereIntersectionTest(geoms[i],r,intersecP,normal);
		else if(geoms[i].type == CUBE ) d = boxIntersectionTest(geoms[i],r,intersecP,normal);

		if( d > 0 && d < min_d )
		{
			min_d = d;
			closestGeomIndex = i; 
		}
	}
	return closestGeomIndex;
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void Accumulate(glm::vec2 resolution, glm::vec3* colors, float iterations,  glm::vec3* current)
//__global__ void Accumulate(glm::vec2 resolution, glm::vec3* colors, float iterations)
{//accumulate color
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	colors[index] = ( colors[index] + current[index] * ( iterations- 1 ) ) / iterations; // / iterations;
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors, 
                            staticGeom* geoms, int numberOfGeoms, material* materials, ray* rays, int numOfRays)//Added cudaMaterial
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);//pixel

	if(x <= cam.resolution.x && y <= cam.resolution.y)
	{//ray r = raycastFromCameraKernel(resolution,time,x,y,cam.position,cam.view,cam.up,cam.fov);
		ray r = rays[index]; 

		glm::vec3 colResult;

		int hitCounter = 0;
		if(rayDepth == 1 )	colors[r.pixelId] = glm:: vec3(1,1,1);

		if( index > numOfRays || r.continueFlag==false ) return;
		glm::vec3 intersecP, norm;
		float d = -1;//distance to intersection
		float min_d = 1000000.0;//the distance to closest object
		glm::vec3 lightPos;
		float lightEmi;
		glm::vec3 lightCol;
		int lightGeoIndex=-1;//store the index of the light source, Only one currently
		float amibient = 0.8, diffuse = 0.8, specular = 0.1;
		
		glm::vec3 emittedColor;
		glm::vec3 unabsorbedColor;
		AbsorptionAndScatteringProperties AbsorpASP;

		int closestGeomIndex = getClosestGeom(geoms, numberOfGeoms, r, intersecP, norm);
		
		//intersection occurred
		if(closestGeomIndex >= 0 )
		{
			material geoMat =  materials[ geoms[closestGeomIndex].materialid ];
			glm::vec3 geoCol;
			//colors[r.pixelId] = glm::vec3(1,1,1);

			// Light
			if( geoMat.emittance > 0) 
			{
				colResult = geoMat.color * geoMat.emittance;// the light color
				r.continueFlag = false;//don't need to keep going
			}

			//Non - Light Objects
			else
			{
				geoCol = geoMat.color;
				thrust::default_random_engine rng(hash(index*time*rayDepth));
				thrust::uniform_real_distribution<float> X1(0, 1);
				float xi1 = X1(rng);
				float xi2 = X1(rng);
				calculateBSDF(r,intersecP, norm, emittedColor, AbsorpASP, colResult, unabsorbedColor, geoMat, xi1, xi2,closestGeomIndex);
			}//END of the intersected object is not light source
		
		}//END of intersect with object

		//NO INTERSECTION
		else
		{
			colResult = glm::vec3(0,0,0);
			r.continueFlag = false;
		}

		colors[r.pixelId] *= colResult;
		rays[index] = r;

		if(rayDepth + 1 > traceDepth && r.continueFlag)
		{
			colors[r.pixelId] = glm:: vec3(0, 0, 0);
			r.continueFlag = false;
		}
	}//this is for the if(x <= cam.resolution.x && y <= cam.resolution.y)
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  //int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  glm::vec3 *current = NULL;

  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);

  hipMalloc((void**)&current, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( current, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  //package materials and sent to GPU
  material* materialsList = new material[numberOfMaterials];
  for(int i=0; i<numberOfMaterials; i++){
	  material newStaticMaterial;
	  newStaticMaterial.color = materials[i].color;
	  newStaticMaterial.specularExponent = materials[i].specularExponent;
	  newStaticMaterial.specularColor = materials[i].specularColor;
	  newStaticMaterial.hasReflective = materials[i].hasReflective;
	  newStaticMaterial.hasRefractive = materials[i].hasRefractive;
	  newStaticMaterial.indexOfRefraction = materials[i].indexOfRefraction;
	  newStaticMaterial.hasScatter = materials[i].hasScatter;
	  newStaticMaterial.absorptionCoefficient = materials[i].absorptionCoefficient;
	  newStaticMaterial.reducedScatterCoefficient = materials[i].reducedScatterCoefficient;
	  newStaticMaterial.emittance = materials[i].emittance;
	  materialsList[i] = newStaticMaterial;
  }

  material* cudaMaterials = NULL;
  hipMalloc((void**)&cudaMaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudaMaterials, materialsList, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

 //Package rays
 int numOfRays = cam.resolution.x * cam.resolution.y;
 ray *rays = new ray[numOfRays];
 ray *cudarays = NULL;
 hipMalloc((void**)&cudarays, numOfRays * sizeof(ray));
 hipMemcpy(cudarays, rays, numOfRays * sizeof(ray), hipMemcpyHostToDevice);
 //////////////////////////////////////////////Focused on the green sphere's Z
 float focus =17;
 generateRay<<<fullBlocksPerGrid, threadsPerBlock>>>(cudarays,cam,iterations,focus);
 //kernel launches
 //traceDepth = 10;

 dim3 blocksPerGrid = fullBlocksPerGrid;

 
 for(int i=1; i < traceDepth + 1; ++i)
 {
	 //raytraceRay(resolution, time, cam, rayDepth, colors,  geoms, numberOfGeoms, materials, rays, numOfRays)
	 raytraceRay<<<blocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, i, cudaimage, cudageoms, numberOfGeoms,cudaMaterials,cudarays,numOfRays);
	 thrust::device_ptr<ray> firstRays_ptr(cudarays);
	 thrust::device_ptr<ray> lastRays_ptr = thrust::remove_if( firstRays_ptr, firstRays_ptr + numOfRays, rayContinueFalse());
	 numOfRays = lastRays_ptr.get() - firstRays_ptr.get();
	 blocksPerGrid = dim3( (int)ceil(renderCam->resolution.x/tileSize), (int)ceil(renderCam->resolution.x)/tileSize);
 }
  Accumulate<<<fullBlocksPerGrid, threadsPerBlock>>>( renderCam->resolution, cudaimage, iterations, current);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
	hipFree( cudaimage );
	hipFree( current );
	hipFree( cudageoms );
	hipFree( cudaMaterials );
	hipFree( cudarays );


  delete geomList;
  delete materialsList;
  delete rays;

  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
